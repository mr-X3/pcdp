#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

__global__ void arrayadd(int *a,int *b,int *c){

	int row=threadIdx.y;
	int col=threadIdx.x;


	c[2*row+col]=a[2*row+col]+b[2*row+col];

}


int main()
{
int size=4;


int a[size],b[size],c[size];


int *h_a,*h_b,*h_c;


for(int i=0;i<size;i++)
{
	a[i]=i*8;
	b[i]=i*5;
	c[i]=0;
}




int gpu_size=sizeof(int)*size;


hipMalloc((void**)&h_a,gpu_size);
hipMalloc((void**)&h_b,gpu_size);
hipMalloc((void**)&h_c,gpu_size);


hipMemcpy(h_a,a,gpu_size,hipMemcpyHostToDevice);
hipMemcpy(h_b,b,gpu_size,hipMemcpyHostToDevice);



arrayadd<<<1,4>>>(h_a,h_b,h_c);



hipMemcpy(c,h_c,gpu_size,hipMemcpyDeviceToHost);


cout<<"Matrix A\n";
for(int i=1;i<=size;i++)
{

	cout<<a[i-1]<<"\t";

	if(i%2==0)
	cout<<"\n";

}

cout<<"Matrix B\n";
for(int i=1;i<=size;i++)
{

	cout<<b[i-1]<<"\t";

	if(i%2==0)
	cout<<"\n";

}



cout<<"Addition is Matrix C\n";
for(int i=1;i<=size;i++)
{

	cout<<c[i-1]<<"\t";

	if(i%2==0)
	cout<<"\n";

}


}

