#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

__global__ void arrayadd(int *a,int *c){

	int tid=threadIdx.x;

	if(tid<100)
	{
		c[tid]=a[tid]*a[tid];
	}

}


int main()
{
int size=100;


int a[size],c[size];


int *h_a,*h_c;


for(int i=0;i<size;i++)
{
	a[i]=i*8;
	c[i]=0;
}




int gpu_size=sizeof(int)*size;


hipMalloc((void**)&h_a,gpu_size);
hipMalloc((void**)&h_c,gpu_size);



hipMemcpy(h_a,a,gpu_size,hipMemcpyHostToDevice);



arrayadd<<<1,1024>>>(h_a,h_c);



hipMemcpy(c,h_c,gpu_size,hipMemcpyDeviceToHost);



cout<<"Array_Square is \n";
for(int i=0;i<size;i++)
{

	cout<<a[i]<<" * "<<a[i]<<" = "<<c[i]<<"\n";
}


}
