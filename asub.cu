#include <hip/hip_runtime.h>

#include <iostream>
using namespace std;

__global__ void arrayadd(int *a,int *b,int *c){

	int tid=threadIdx.x;

	if(tid<100)
	{
		c[tid]=a[tid]-b[tid];
	}

}


int main()
{
int size=100;


int a[size],b[size],c[size];


int *h_a,*h_b,*h_c;


for(int i=0;i<size;i++)
{
	a[i]=i*8;
	b[i]=i*7;
	c[i]=0;
}




int gpu_size=sizeof(int)*size;


hipMalloc((void**)&h_a,gpu_size);
hipMalloc((void**)&h_b,gpu_size);
hipMalloc((void**)&h_c,gpu_size);



hipMemcpy(h_a,a,gpu_size,hipMemcpyHostToDevice);
hipMemcpy(h_b,b,gpu_size,hipMemcpyHostToDevice);



arrayadd<<<1,1024>>>(h_a,h_b,h_c);



hipMemcpy(c,h_c,gpu_size,hipMemcpyDeviceToHost);




for(int i=0;i<size;i++)
{

	cout<<a[i]<<" - "<<b[i]<<" = "<<c[i]<<"\n";
}


}

