
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define BLOCK_SIZE 16

__global__ void gpuMM(int *A, int *B, int *C, int N)
{

	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	int sum = 0;
	for (int n = 0; n < N; ++n)
	    sum += A[row*N+n]*B[n*N+col];

	C[row*N+col] = sum;
}

int main(int argc, char *argv[])
{

	int N=2;

	cout << "Executing Matrix Multiplcation" << endl;
	cout << "Matrix size: " << N << "x" << N << endl;


	int hA[N*N],hB[N*N],hC[N*N];






	for (int j=0; j<N*N; j++){

			hA[j]=j;
			hB[j]=j;
			hC[j]=0;


	}


	int size = N*N*sizeof(int);

	int *dA,*dB,*dC;

	hipMalloc((void**)&dA,size);
	hipMalloc((void**)&dB,size);
	hipMalloc((void**)&dC,size);

	dim3 threadBlock(4,4);
	dim3 grid(1,1);


	hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
	hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);


	gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);


	hipMemcpy(hC,dC,size,hipMemcpyDeviceToHost);



	cout<<"Matrix A\n";
		for(int i=1;i<=N*N;i++)
		{

			cout<<hA[i-1]<<"\t";

			if(i%2==0)
			cout<<"\n";

		}


		cout<<"Matrix B\n";
			for(int i=1;i<=N*N;i++)
			{

				cout<<hB[i-1]<<"\t";

				if(i%2==0)
				cout<<"\n";

			}



	cout<<"Matrix C is \n";
	for(int i=1;i<=N*N;i++)
	{

		cout<<hC[i-1]<<"\t";

		if(i%2==0)
		cout<<"\n";

	}



	cout << "Finished." << endl;

}

